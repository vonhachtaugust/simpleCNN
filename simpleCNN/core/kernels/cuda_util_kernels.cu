#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cuda_util_kernels.h"

///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
__global__ void FillOnes(float *vec, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;

    vec[idx] = 1.0f;
}

float_t* onevec() {

    return nullptr;
}